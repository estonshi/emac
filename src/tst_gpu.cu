#include <stdio.h>

#include "hip/hip_runtime.h"


int main(){
    hipError_t custatus;
    int gpu_num = 0;
    hipDeviceProp_t prop;
    
    custatus = hipGetDeviceCount(&gpu_num);
    printf("Number of GPUs       : %d\n", gpu_num);

    custatus = hipSetDevice(0);
    if(custatus != hipSuccess){
        printf("Failed to set Device 0. Exit\n");
        return -1;
    }

    hipGetDeviceProperties(&prop, 0);
    printf("Device name          : %s\n", prop.name);
    printf("Compute Capability   : %d.%d\n", prop.major, prop.minor);
    printf("totalGlobalMem       : %.1f GB\n", prop.totalGlobalMem/1024/1024/1024.0);
    printf("sharedMemPerBlock    : %u KB\n", (unsigned int)prop.sharedMemPerBlock/1024);
    printf("maxThreadsPerBlock   : %d\n", prop.maxThreadsPerBlock);
    printf("maxGridSize          : %d, %d, %d\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("regPerBlock          : %d\n", prop.regsPerBlock);
    
}
